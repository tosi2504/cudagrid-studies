#include "cudatools.h"
#include "sgemm.h"
#include "matrix.h"
#include <chrono>
#include <iostream>

decltype(std::chrono::high_resolution_clock::now()) start;
decltype(std::chrono::high_resolution_clock::now()) stop;

inline void timing_start() {
	start = std::chrono::high_resolution_clock::now();
}

inline long timing_stop() {
	stop = std::chrono::high_resolution_clock::now();
	auto delta_t = std::chrono::duration_cast<std::chrono::microseconds>(stop - start); // fix type issue
	return delta_t.count();
}

constexpr unsigned reps = 50;
constexpr unsigned long N = 2048;
constexpr uint BN = 128;
static_assert(N%BN == 0);
constexpr uint BK = 8;
static_assert(N%BK == 0);
constexpr uint TN = 8;
using T = float;
int main () {
	Matrix<T> A(N), B(N), C(N);
	A.fill_random(0);
	A.upload();
	B.fill_random(1);
	B.upload();
	C.fill_random(2);
	C.upload();


	dim3 gridDim(N/BN, N/BN, 1);
	dim3 blockDim((BN*BN)/(TN*TN), 1, 1);
	std::cout << "USING " << (BN*BN)/(TN*TN) << " THREADS PER BLOCK" << std::endl;

	float alpha = 1;
	float beta = 0;
	std::cout << "STARTED TIMING" << std::endl;
	timing_start();
	for (unsigned rep = 0; rep < reps; rep++) {
		sgemm_vectorized<float, N, BN, BK, TN> <<< gridDim , blockDim >>> (C.d_data, A.d_data, B.d_data, alpha, beta);
		CLCE();
		CCE(hipDeviceSynchronize());
	}
	unsigned microsecs = timing_stop();
	std::cout << "BANDWIDTH (MByte/s): " << sizeof(float)*reps*(4*N*N)/(double)microsecs << std::endl;
	std::cout << "ARITHETICS (GFLOPS/s) (TODO): " << reps*(2*N*N*N + N*N)/((double)microsecs*1000) << std::endl;
	CLCE();

	// check for correctness
	C.download();
	uint i = N-1;
	uint j = N-1;
	std::cout << "i: " << i << " j: " << j << std::endl;
	std::cout << Matrix<T>::matmul(A, B, i, j) << " <---> " << C.get(i,j) << std::endl;
	// checkMatmul(C, A, B);
}
